#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#define CUSTOMMERS 400
#define PEOPLE_PER_TABLE 10
#define WAITERS_PER_TABLE 2
#define TABLES CUSTOMMERS/PEOPLE_PER_TABLE

using namespace std;

__global__ void GPU_serving_dishes(int round_offset, bool* plates) 
{
	int gId = threadIdx.x + round_offset;
	plates[gId] = true;
}

__host__ void CPU_serving_dishes() 
{
	int serving_rounds = CUSTOMMERS / (TABLES * WAITERS_PER_TABLE);

	bool* host_customers_plate;
	bool* dev_customers_plate;

	host_customers_plate = (bool*)malloc(CUSTOMMERS * sizeof(bool));
	hipMalloc((void**)&dev_customers_plate, CUSTOMMERS * sizeof(bool));

	for (int i = 0; i < CUSTOMMERS; i++)
	{
		host_customers_plate[i] = false;
	}
	for (int i = 0; i < serving_rounds; i++)
	{
		hipMemcpy(dev_customers_plate, host_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyHostToDevice);
		dim3 block(CUSTOMMERS / serving_rounds);
		GPU_serving_dishes << <1, block >> > (i * CUSTOMMERS / serving_rounds, dev_customers_plate);
		hipDeviceSynchronize();
		hipMemcpy(host_customers_plate, dev_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyDeviceToHost);

		float percentage_served = 0;
		int persons_served = 0;
		for (int i = 0; i < CUSTOMMERS; i++)
		{
			if (host_customers_plate[i]) {
				persons_served++;
			}
		}
		percentage_served = 100 * persons_served / (float)CUSTOMMERS;
		printf("Persons served: %.2f%%\n", percentage_served);

	}

	free(host_customers_plate);
	hipFree(dev_customers_plate);
}

__host__ float getRangeRandom()
{
	return (rand() % 11 / 10.0) * 10;
}


__host__ void fillRandomNumbersList(float* randomNumbersList, int size)
{
	for (int i = 0; i < size; i++)
	{
		randomNumbersList[i] = getRangeRandom();
	}
}

__host__ void CPU_cleaning_table_and_picking_up_plates() {
	int serving_rounds = CUSTOMMERS / (TABLES * WAITERS_PER_TABLE);

	bool* host_customers_plate;
	bool* dev_customers_plate;
	float* host_random_eating_times;
	float* dev_random_eating_times;

	host_customers_plate = (bool*)malloc(CUSTOMMERS * sizeof(bool));
	host_random_eating_times = (float*)malloc(CUSTOMMERS * sizeof(float));
	hipMalloc((void**)&dev_customers_plate, CUSTOMMERS * sizeof(bool));
	hipMalloc((void**)&dev_random_eating_times, CUSTOMMERS * sizeof(float));

	for (int i = 0; i < CUSTOMMERS; i++)
	{
		host_customers_plate[i] = false;
	}
	fillRandomNumbersList(host_random_eating_times, CUSTOMMERS);

	hipMemcpy(dev_customers_plate, host_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyHostToDevice);

	free(host_customers_plate);
	hipFree(dev_customers_plate);
}
int main() {
	printf("START\n");

	printf("Cooking food...\n");
	_sleep(1000);
	printf("Food Cooked\n");

	printf("Serving food...\n");
	CPU_serving_dishes();
	printf("Food served\n");
	/*
	printf("People Eating...\n");
	CPU_cleaning_table_and_picking_up_plates();
	printf("Some People Finishing\n");
	*/
	return 0;
}