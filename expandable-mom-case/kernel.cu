#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#define CUSTOMMERS 400
#define PEOPLE_PER_TABLE 10
#define WAITERS_PER_TABLE 2
#define TABLES CUSTOMMERS/PEOPLE_PER_TABLE

using namespace std;

// GPU function that simulates an sleep function
__device__ void wait_gpu(float time) {
	int wait = 1000000 * time * 2.5;
	for (int i = 0; i < wait; i++)
	{
	}
}

// GPU function that simulates the serving dishes time
__global__ void GPU_serving_dishes(int round_offset, bool* plates) 
{
	int gId = threadIdx.x + round_offset;
	wait_gpu(1);
	plates[gId] = true;
}

// GPU function that simulates the eating and picking plates up time
__global__ void GPU_finish_eating_and_picking_up_plates(bool* plates, float* eating_times, clock_t* global_now)
{
	int gId = threadIdx.x;
	wait_gpu(eating_times[gId]);
	wait_gpu(1);
	plates[gId] = true;
}

// Simulates how the catering service will behave in the event
__host__ void CPU_serving_dishes(char food_course[], hipEvent_t s, hipEvent_t e)
{
	printf("SERVICE: serving %s ...\n", food_course);

	// Determine how many rounds did the waiters will need to do, to serve a table
	int serving_rounds = CUSTOMMERS / (TABLES * WAITERS_PER_TABLE);

	// Declare host and device variables
	bool* host_customers_plate;
	bool* dev_customers_plate;

	// Reserve space for host and device variables
	host_customers_plate = (bool*)malloc(CUSTOMMERS * sizeof(bool));
	hipMalloc((void**)&dev_customers_plate, CUSTOMMERS * sizeof(bool));

	// Initialice host variables
	for (int i = 0; i < CUSTOMMERS; i++)
	{
		host_customers_plate[i] = false;
	}
	float totalTime = 0;

	// Start serving rounds
	for (int i = 0; i < serving_rounds; i++)
	{
		hipMemcpy(dev_customers_plate, host_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyHostToDevice);
		dim3 block(CUSTOMMERS / serving_rounds);

		// Start kernel function and capture the time it spend in the execution
		hipEventCreate(&s);
		hipEventCreate(&e);
		hipEventRecord(s, 0);
		GPU_serving_dishes << <1, block >> > (i * CUSTOMMERS / serving_rounds, dev_customers_plate);
		hipEventRecord(e, 0);
		hipDeviceSynchronize();
		float currElapsedTime;
		hipEventElapsedTime(&currElapsedTime, s, e);
		totalTime += currElapsedTime;
		hipEventDestroy(s);
		hipEventDestroy(e);
		
		hipMemcpy(host_customers_plate, dev_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyDeviceToHost);

		// Calculate the percentage of people that have been served
		float percentage_served = 0;
		int persons_served = 0;
		for (int i = 0; i < CUSTOMMERS; i++)
		{
			if (host_customers_plate[i]) {
				persons_served++;
			}
		}
		percentage_served = 100 * persons_served / (float)CUSTOMMERS;
		printf("SERVICE: people served -> %.2f%%\n", percentage_served);

	}

	// Free space for host and device variables
	free(host_customers_plate);
	hipFree(dev_customers_plate);
	printf("SERVICE: %s served in %.3f ms\n", food_course, totalTime);
}

// Generates a random number from 0-10 
__host__ float getRangeRandom()
{
	return (rand() % 11 / 10.0) * 10;
}

// Fills a float list with random numbers
__host__ void fillRandomNumbersList(float* randomNumbersList, int size)
{
	for (int i = 0; i < size; i++)
	{
		randomNumbersList[i] = getRangeRandom()+1;
	}
}

// Simulates the eating and picking up of the plates when a costummer has finish
__host__ void CPU_finish_eating_and_picking_up_plates(hipEvent_t s, hipEvent_t e) {
	printf("FOOD: people eating...\n");
	int serving_rounds = CUSTOMMERS / (TABLES * WAITERS_PER_TABLE);

	// Declare host and device variables
	bool* host_customers_plate;
	bool* dev_customers_plate;
	float* host_random_eating_times;
	float* dev_random_eating_times;
	clock_t* host_global_clock;
	clock_t* dev_global_clock;

	// Reserve space for host and device variables
	host_customers_plate = (bool*)malloc(CUSTOMMERS * sizeof(bool));
	host_random_eating_times = (float*)malloc(CUSTOMMERS * sizeof(float));
	host_global_clock = (clock_t*)malloc(sizeof(clock_t));
	hipMalloc((void**)&dev_customers_plate, CUSTOMMERS * sizeof(bool));
	hipMalloc((void**)&dev_random_eating_times, CUSTOMMERS * sizeof(float));
	hipMalloc((void**)&dev_global_clock, sizeof(clock_t));

	// Initialice host variables
	for (int i = 0; i < CUSTOMMERS; i++)
	{
		host_customers_plate[i] = false;
	}
	fillRandomNumbersList(host_random_eating_times, CUSTOMMERS);

	hipMemcpy(dev_customers_plate, host_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(dev_random_eating_times, host_random_eating_times, CUSTOMMERS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_global_clock, host_global_clock, sizeof(clock_t), hipMemcpyHostToDevice);
	dim3 block(CUSTOMMERS);

	// Start kernel function and capture the time it spend in the execution
	float totalTime = 0;
	hipEventCreate(&s);
	hipEventCreate(&e);
	hipEventRecord(s, 0);
	GPU_finish_eating_and_picking_up_plates << < 1, block >> > (dev_customers_plate,dev_random_eating_times, dev_global_clock);
	hipEventRecord(e, 0);
	hipDeviceSynchronize();
	float currElapsedTime;
	hipEventElapsedTime(&currElapsedTime, s, e);
	totalTime += currElapsedTime;
	hipEventDestroy(s);
	hipEventDestroy(e);

	hipMemcpy(host_customers_plate, dev_customers_plate, CUSTOMMERS * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(host_random_eating_times, dev_random_eating_times, CUSTOMMERS * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(host_global_clock, dev_global_clock, sizeof(clock_t), hipMemcpyDeviceToHost);

	// Free space for host and device variables
	free(host_customers_plate);
	free(host_random_eating_times);
	free(host_global_clock);
	hipFree(dev_customers_plate);
	hipFree(dev_random_eating_times);
	hipFree(dev_global_clock);
	printf("FOOD: people finish eating and plates picked up in %.3f ms\n", currElapsedTime);
}

int main() {
	printf("START\n");

	printf("SERVICE: Preparing food...\n");
	_sleep(1000);
	printf("SERVICE: Food ready\n");

	// Starting cuda events, these help us calculating the time spend in the kernel
	hipEvent_t start;
	hipEvent_t end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	// Three course dinner
	CPU_serving_dishes("starter", start, end);
	CPU_finish_eating_and_picking_up_plates(start, end);
	CPU_serving_dishes("main course", start, end);
	CPU_finish_eating_and_picking_up_plates(start, end);
	CPU_serving_dishes("dessert", start, end);
	CPU_finish_eating_and_picking_up_plates(start, end);

	printf("END\n");
	return 0;
}